#include <stdio.h>
#include <stdlib.h>
#include <hip/hip_runtime.h>



__device__ bool collatzConverge(uint64_t x) {
    // Executa a sequência Collatz em registradores locais até:
    //  a) x == 1    → retorna true (convergiu)
    //  b) x < start → retorna true (já caiu em intervalo testado)
    //  c) ou até um limite máximo de iterações → retorna false (suspenso)
    // Para detectar ciclo, opcionalmente podia guardar últimos K valores, mas isso complica.
    uint64_t orig = x;
    int maxIter = 10000;  // limite arbitrário para cada número
    for (int i = 0; i < maxIter; i++) {
        if ((x & 1) == 0) {
            x = x >> 1;
        } else {
            x = 3*x + 1;
        }
        if (x == 1 || x < orig) {
            return true;  // “convergiu” para 1 ou entrou em intervalo já testado
        }
        // Note: se x crescer > 2^63, overflow vai acontecer, mas ignoramos
        // Se quiser detectar overflow, checar antes de 3*x+1:
        // if (x > (UINT64_MAX-1)/3) return false; 
    }
    return false;  // excedeu maxIter sem convergir → tratamos como “não investigado totalmente”
}

__global__ void checkCollatzKernel(uint64_t startN, uint64_t range, int *flag) {
    // Cada thread testa um único valor de n:
    uint64_t idx = blockIdx.x * blockDim.x + threadIdx.x;
    if (idx >= range) return;
    uint64_t n = startN + idx;  // mapeia “idx” → valor natural
    if (!collatzConverge(n)) {
        // Usuario finalizador: se encontrou candidato “suspeito”, grava índice
        atomicExch(flag, 1);
    }
}


int main() {
    uint64_t faixaSize = 50'000'000;   // testamos 50M de cada vez (ajustável)
    uint64_t currentStart = 1;
    int *d_flag;
    int h_flag;

    hipMalloc(&d_flag, sizeof(int));

    while (true) {
        // (1) Resetar flag
        h_flag = 0;
        hipMemcpy(d_flag, &h_flag, sizeof(int), hipMemcpyHostToDevice);

        // (2) Lançar kernel para essa faixa [currentStart .. currentStart + faixaSize - 1]
        int TPB = 256;
        int numBlocks = (faixaSize + TPB - 1) / TPB;
        checkCollatzKernel<<<numBlocks, TPB>>>(currentStart, faixaSize, d_flag);
        hipDeviceSynchronize();

        // (3) Ler flag de volta
        hipMemcpy(&h_flag, d_flag, sizeof(int), hipMemcpyDeviceToHost);
        if (h_flag != 0) {
            printf("Encontrado candidato a contraexemplo na faixa [%llu .. %llu]\n",
                   currentStart, currentStart + faixaSize - 1);
            break;
        }

        // (4) Avançar para próxima faixa
        currentStart += faixaSize;
        printf("Faixa [%llu .. %llu] processada—continuando...\n",
               currentStart - faixaSize, currentStart - 1);

        // (Opcional) critério de parada extra para não rodar eternamente:
        // if (currentStart > LIMITE_SUPERIOR) break;
    }

    hipFree(d_flag);
    return 0;
}
